#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdlib.h>
#include <stdio.h>
#include <memory.h>
#include <cmath>
#include <sys/time.h>
#include <stdint.h>

#define USECPMSEC 1000ULL
#define USECPSEC 1000000ULL

void matrix_init_rand(float * matrix, int x, int y);
__global__ void MatrixMulKernel(float* d_M, float* d_N, float* d_P, int M_row, int Mul_line, int N_col);
__global__ void MatrixTiledMulKernel(float* d_M, float* d_N, float* d_P, int M_row, int MUL_line, int N_col, int TILE_WIDTH);
void print_matrix(float * matrix, int x, int y);
uint64_t dtime_usec(uint64_t start);


int main (int argc, char **argv){
    srand((unsigned)time(NULL) );
    int M_row = 4; int MUL_line = 4; int N_col = 4;
    int TILE_WIDTH = 1;
    int EQUAL_MODE = 0;
    int TILE_CACHING_MODE = 0;
    int PRINT_MATRIX_MODE = 0;

    int param_num = 0;

    int param_opt;
    while(-1 !=(param_opt = getopt(argc, argv, "ntp"))){

	    switch(param_opt){
	        case 'n' : 
                EQUAL_MODE = 1;
                param_num ++;
                break;
            case 't' : 
                TILE_CACHING_MODE = 1;
                param_num ++;
                break;
            case 'p' :
                PRINT_MATRIX_MODE = 1;
                param_num ++;
                break;
		}
        
	}

    if(EQUAL_MODE ==0){
        M_row = atoi(argv[1+param_num]);
        MUL_line = atoi(argv[2+param_num]);
        N_col = atoi(argv[3+param_num]);
        if(argc ==5+param_num) TILE_WIDTH = atoi(argv[4+param_num]);
    }
    else{ //EQUAL_MODE ==1
        M_row = atoi(argv[1+param_num]);
        MUL_line = M_row;
        N_col = M_row;
        if(argc ==3+param_num) TILE_WIDTH = atoi(argv[2+param_num]);
    }

    float * M, *N, *P; // MN = P
    hipHostAlloc((void**)&M, sizeof(float)*M_row*MUL_line, hipHostMallocDefault);
    hipHostAlloc((void**)&N, sizeof(float)*MUL_line*N_col, hipHostMallocDefault);
    hipHostAlloc((void**)&P, sizeof(float)*M_row*N_col, hipHostMallocDefault);

    matrix_init_rand(M, M_row,MUL_line);
    matrix_init_rand(N, MUL_line,N_col);
    memset(P, 0.0, sizeof(float)*M_row*N_col);

    float * cu_M, * cu_N, * cu_P;
    hipMalloc((void**)&cu_M, sizeof(float)*M_row*MUL_line);
    hipMalloc((void**)&cu_N, sizeof(float)*MUL_line*N_col);
    hipMalloc((void**)&cu_P, sizeof(float)*M_row*N_col);
    
    hipMemcpy(cu_M, M, sizeof(float)*M_row*MUL_line, hipMemcpyHostToDevice);
    hipMemcpy(cu_N, N, sizeof(float)*MUL_line*N_col, hipMemcpyHostToDevice);
    hipMemcpy(cu_P, P, sizeof(float)*M_row*N_col, hipMemcpyHostToDevice);

    if(PRINT_MATRIX_MODE ==1){
        printf("M Matrix:::\n");
        print_matrix(M, MUL_line, M_row);
        printf("\nN Matrix:::\n");
        print_matrix(N, N_col, MUL_line);
        printf("\n\n");
    }
    hipHostFree(M); hipHostFree(N);

    uint64_t difft = dtime_usec(0);
    if(TILE_CACHING_MODE ==1){
        dim3 dimGrid( ceil(N_col/(TILE_WIDTH*1.0)), ceil(M_row/(TILE_WIDTH*1.0)), ceil(MUL_line/(TILE_WIDTH*1.0)) );
        dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
        MatrixTiledMulKernel<<<dimGrid, dimBlock, sizeof(float)*TILE_WIDTH*TILE_WIDTH*2>>>(cu_M, cu_N, cu_P, M_row, MUL_line, N_col, TILE_WIDTH);

    }
    else if(TILE_CACHING_MODE ==0){
        dim3 dimGrid(ceil(N_col/(TILE_WIDTH*1.0)),  ceil(M_row/(TILE_WIDTH*1.0)), 1 );
        dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
        MatrixMulKernel<<<dimGrid,dimBlock>>>(cu_M, cu_N, cu_P, M_row, MUL_line, N_col);
    }
    hipDeviceSynchronize();
    difft = dtime_usec(difft);

    hipMemcpy(P, cu_P, sizeof(float)*M_row*N_col, hipMemcpyDeviceToHost);

    if(PRINT_MATRIX_MODE ==1){
        printf("P Matrix:::\n");
        print_matrix(P, N_col, M_row);
    }
    else{ //print time only print matrix mode is deactivated.
        printf("GPU COMPUTE TIME(ms): %f\n", difft/(float)USECPMSEC);
    }
    
    hipHostFree(P); hipFree(cu_M); hipFree(cu_N); hipFree(cu_P); 
    
    hipError_t err = hipGetLastError();
    if(hipSuccess != err)
        printf("*cudaErr(%d): %s \n",err,hipGetErrorString(err));
    return 0;
}

__global__ void MatrixTiledMulKernel(float* d_M, float* d_N, float* d_P, int M_row, int MUL_line, int N_col, int TILE_WIDTH)
{
    int local_col = threadIdx.x;
    int local_row = threadIdx.y;

    int Z = blockIdx.z*blockDim.z;
    int Col = blockIdx.x*blockDim.x+local_col;
    int Row = blockIdx.y*blockDim.y+local_row;

    extern __shared__ float sharedCache[];
    
    //캐시에 데이터 업로드
    int local_index = TILE_WIDTH*local_row + local_col;
    int arr_jump = TILE_WIDTH * TILE_WIDTH;
    int global_col = (Z*TILE_WIDTH) +local_col;
    int global_row = (Z*TILE_WIDTH) + local_row;


    sharedCache[local_index] = d_M[MUL_line * (Row) + global_col];
    sharedCache[arr_jump + local_index] = d_N[N_col* global_row + Col];

    __syncthreads();

    float Pvalue = 0;
    if ((Row < M_row) && (Col < N_col)) {

        for (int k = 0; k < TILE_WIDTH; ++k){

            if( (global_col + k > N_col) || (global_row + k > M_row)) break; 

            Pvalue += sharedCache[ local_row*TILE_WIDTH +k ] * sharedCache[ arr_jump + k*TILE_WIDTH + local_col];           
        }
    }

    atomicAdd(&d_P[Row*N_col+Col], Pvalue);
}

__global__ void MatrixMulKernel(float* d_M, float* d_N, float* d_P, int M_row, int Mul_line, int N_col)
{
    // Calculate the row index of the d_P element and d_M
    int Row = blockIdx.y*blockDim.y+threadIdx.y;
    // Calculate the column idenx of d_P and d_N
    int Col = blockIdx.x*blockDim.x+threadIdx.x;
    if ((Row < M_row) && (Col < N_col)) {
        float Pvalue = 0;
        // each thread computes one element of the block sub-matrix
            for (int k = 0; k < Mul_line; ++k)
                Pvalue += d_M[Row*Mul_line+k] * d_N[k*N_col+Col];
            d_P[Row*N_col+Col] = Pvalue;
    }
}


void matrix_init_rand(float * matrix, int x, int y){
    int size = x*y;
	for(int i=0;i<size;i++){
		matrix[i] = (float) ((rand()%1000) -500) / 100.0 ;
		//matrix[i] = i; //for debug
	}
}

void print_matrix(float * matrix, int x, int y){
	for (int i=0;i<y;i++){
		for (int j=0;j<x;j++){
			printf("%f ", matrix[i*x + j]);	
		}
		printf("\n");
	}
}

uint64_t dtime_usec(uint64_t start)
{
  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}
