

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cmath>

#define KERNEL_SIZE 3   // change this to kernel size
#define TILE_SIZE 5
#define BLOCK_SIZE ( TILE_SIZE + (KERNEL_SIZE-1) )

__constant__ float Mc[KERNEL_SIZE][KERNEL_SIZE][KERNEL_SIZE];

__global__ 
void MatrixConv(float* N, float* P, int x, int y, int z)
{   
    int tx = threadIdx.x, ty = threadIdx.y, tz = threadIdx.z; 
    int y_o = blockIdx.y*TILE_SIZE + ty;
    int x_o = blockIdx.x*TILE_SIZE + tx;
    int z_o = blockIdx.z*TILE_SIZE + tz;

    int move_size = (KERNEL_SIZE -1) / 2;
    int x_i = x_o - move_size, y_i = y_o - move_size, z_i = z_o - move_size;

    float output = 0.0f;
    __shared__ float Ns[TILE_SIZE+KERNEL_SIZE-1][TILE_SIZE+KERNEL_SIZE-1][TILE_SIZE+KERNEL_SIZE-1];


    if((x_i >= 0) && (x_i < x) && (y_i >= 0) && (y_i < y) && (z_i >= 0) && (z_i < z)) {
        Ns[tz][ty][tx] = N[z_i*y*x + y_i*x + x_i];
    }
    else {
        Ns[tz][ty][tx] = 0.0f;
    }
    __syncthreads();
    
    if(ty < TILE_SIZE && tx < TILE_SIZE && tz < TILE_SIZE){
        for(int k = 0; k < KERNEL_SIZE; k++) {
            for(int i = 0; i < KERNEL_SIZE; i++) {
                for(int j = 0; j < KERNEL_SIZE; j++) {
                    output += Mc[k][i][j] * Ns[k+tz][i+ty][j+tx];
                }
            }
        }
        // some threads do not write output
        if (y_o < y && x_o < x && z_o < z) {
            P[z_o*y*x + y_o*x + x_o] = output;
        }
    }
    
}

void verification(const float *P, const char* sample_file) {
    int x, y, z, idx=0;
    float n;
    bool equal = true;
    FILE *sample = fopen(sample_file, "r");
    fscanf(sample, "%d %d %d", &z, &y, &x);

	while (equal) {
        if(idx == x*y*z) break;
        if(fscanf(sample, "%f ", &n) != 1) {
            printf("Error while verification!\n");
            fclose(sample);
            return;
        }
        if(fabs((double)(P[idx++] - n)) > 0.0001) {
            equal = false;
        }
    }

	if (equal) {
		printf("Results are equal!\n");
	}
	else {
		printf("Results are NOT equal!\n");
	}
    fclose(sample);
	return;
}

int main(int argc, char** argv) {
    float time_ms = 0;
    hipEvent_t t1, t2;
    int z, y, x;
    float n;
    float *h_N, *h_M, *h_P;
    float *N, *P;
    int kernel_size;

    if(argc != 5) {
        fprintf(stderr, "%s", "Execution Error. Command example : \
                './matconv [input] [kernel] [output] [sample]'.\n\n");
        return 1;
    }

    //read input file
    FILE *input = fopen(argv[1], "r");
    fscanf(input, "%d %d %d", &z, &y, &x);
    // printf("z:%d, y:%d, x:%d\n",z ,y, x);

    h_N = (float*)malloc(sizeof(float)* z*y*x);
    int idx=0;
    while (fscanf(input, "%f ", &n) == 1) {
        h_N[idx++] = n;
    }

    //read kernel
    FILE *kernel = fopen(argv[2], "r");
    fscanf(kernel, "%d", &kernel_size);
    // printf("kernel size:%d\n", kernel_size);
    h_M = (float*)malloc(sizeof(float)* kernel_size * kernel_size * kernel_size);
    idx = 0;
    while (fscanf(kernel, "%f ", &n) == 1) {
        h_M[idx++] = n;
    }

    //malloc
    h_P = (float*)malloc(sizeof(float)* z*y*x);
    hipMalloc((void**)&N, sizeof(float) * z*y*x);
    hipMalloc((void**)&P, sizeof(float) * z*y*x);
    hipMemcpy(N, h_N, sizeof(float) * z*y*x, hipMemcpyHostToDevice);
    hipMemset(P, 0, sizeof(float)* z*y*x);
    hipMemcpyToSymbol(HIP_SYMBOL(Mc), h_M, sizeof(float) * kernel_size*kernel_size*kernel_size);

    //execute convolution
    dim3 dimGrid(
        ceil(x/(TILE_SIZE*1.0)), 
        ceil(y/(TILE_SIZE*1.0)), 
        ceil(z/(TILE_SIZE*1.0)));
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);

    //measure time
    hipEventCreate(&t1);
	hipEventCreate(&t2);
    
    hipEventRecord(t1, 0);
    MatrixConv<<<dimGrid, dimBlock>>>(N, P, x, y, z);
    hipEventRecord(t2, 0);

	hipEventSynchronize(t2);
	hipEventElapsedTime(&time_ms, t1, t2);

    hipMemcpy(h_P, P, sizeof(float) * z*y*x, hipMemcpyDeviceToHost);

    //write output file
    FILE *output = fopen(argv[3], "w");
    fprintf(output, "%d %d %d ", z, y, x);
    for(int i=0; i<z; i++) {
        for(int j=0; j<y; j++) {
            for(int k=0; k<x; k++) {
                fprintf(output, "%.6f ", h_P[i*y*x + j*x + k]);
            }
        }
    }
    verification(h_P, argv[4]);
    printf("CUDA GPU execution time : %.5f ms\n\n", time_ms);

    hipFree(N); hipFree(P);
    free(h_N); free(h_M); free(h_P);
    fclose(input); fclose(kernel); fclose(output);

    hipError_t err = hipGetLastError();
    if(hipSuccess != err)
        printf("*cudaErr(%d): %s \n",err,hipGetErrorString(err));
    return 0;
}