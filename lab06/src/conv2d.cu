#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdlib.h>
#include <stdio.h>
#include <memory.h>
#include <cmath>
#include <sys/time.h>
#include <stdint.h>

#define USECPMSEC 1000ULL
#define USECPSEC 1000000ULL
#define TILE_SIZE 10
#define KERNEL_SIZE 3
#define BLOCK_SIZE ( TILE_SIZE + (KERNEL_SIZE-1) )

__constant__ float Mc [KERNEL_SIZE][KERNEL_SIZE]; //커널

__global__ void Conv2D(float* P, float* N, int hegiht, int width);
void verification(const float *N, const float *M, const float *P, int Rows, int Columns);
void matrix_init_rand(float * matrix, int x, int y);
void print_matrix(float * matrix, int x, int y);



int main (int argc, char **argv){
    srand((unsigned)time(NULL) );
    int rows, columns;
    rows = atoi(argv[1]);
    columns = atoi(argv[2]);
    size_t ARRSIZE = sizeof(float)*rows*columns;
    size_t KSIZE = sizeof(float)*KERNEL_SIZE*KERNEL_SIZE;
    
    //호스트 메모리에 인풋, 아웃풋 공간 할당 및 값 입력
    float * intputArr = (float*) malloc(ARRSIZE);
    float * outputArr = (float*) malloc(ARRSIZE);
    matrix_init_rand(intputArr, rows, columns);
    memset(outputArr, 0.0, ARRSIZE);
    
    //GPU 메모리에 인풋, 아웃풋 공간 할당 및 카피
    float * cu_inputArr, * cu_outputArr;
    hipMalloc((void**)&cu_inputArr, ARRSIZE);
    hipMalloc((void**)&cu_outputArr, ARRSIZE);
    hipMemcpy(cu_inputArr, intputArr, ARRSIZE, hipMemcpyHostToDevice);
    hipMemcpy(cu_outputArr, outputArr, ARRSIZE, hipMemcpyHostToDevice);

    //커널 할당 및 값 입력. (최종적으로 __CONST__ 변수 Mc에 할당)
    float * kernel;
    kernel = (float*) malloc(KSIZE);
    matrix_init_rand(kernel, KERNEL_SIZE, KERNEL_SIZE);
    float * cu_temp;
    hipMalloc((void**)&cu_temp, KSIZE);
    hipMemcpy(cu_temp, kernel, KSIZE, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(Mc), cu_temp, KSIZE);
    hipFree(cu_temp);

    //int BLOCK_SIZE = TILE_SIZE + (KERNEL_SIZE – 1);
    dim3 dimGrid( ceil(columns/(TILE_SIZE*1.0)), ceil(rows/(TILE_SIZE*1.0)), 1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    Conv2D<<<dimGrid, dimBlock>>>(cu_outputArr, cu_inputArr, columns, rows);
    hipDeviceSynchronize();
    
    hipMemcpy(outputArr, cu_outputArr, ARRSIZE, hipMemcpyDeviceToHost);

    verification(intputArr, kernel, outputArr, rows, columns);
    //N: 인풋 데이터 //M: 커널 //P: CUDA 연산 결과
    
    free(outputArr); free(intputArr);
    free(kernel);
    hipFree(cu_inputArr); hipFree(cu_outputArr);
    
    hipError_t err = hipGetLastError();
    if(hipSuccess != err)
        printf("*cudaErr(%d): %s \n",err,hipGetErrorString(err));
    return 0;
    
    
}

__global__ void Conv2D(float* P, float* N, int hegiht, int width){
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row_o = blockIdx.y * TILE_SIZE + ty;
    int col_o = blockIdx.x * TILE_SIZE + tx;

    int row_i = row_o - ((KERNEL_SIZE - 1) / 2);
    int col_i = col_o - ((KERNEL_SIZE - 1) / 2);

    float output = 0.0f;
    __shared__ float Ns[TILE_SIZE+KERNEL_SIZE-1][TILE_SIZE+KERNEL_SIZE-1];
    if((row_i >= 0) && (row_i < hegiht) && (col_i >= 0) && (col_i < width))
        Ns[ty][tx] = N[row_i*width + col_i];
    else
        Ns[ty][tx] = 0.0f;

    __syncthreads();
    
    if(ty < TILE_SIZE && tx < TILE_SIZE){
        for(int i = 0; i < KERNEL_SIZE; i++)
            for(int j = 0; j < KERNEL_SIZE; j++)
                output += Mc[i][j] * Ns[i+ty][j+tx];

        if (row_o < hegiht && col_o < width)
        P[row_o * width + col_o] = output;
    }
    
}


void verification(const float *N, const float *M, const float *P, int Rows, int Columns) {
	int r, c, h, w;
	int row_i, col_i;
	bool equal;
	float* results;

	results = (float*)malloc(Rows * Columns * sizeof(float));
	memset(results, 0, Rows * Columns * sizeof(float));

	for (r = 0; r < Rows; r++) {
		for (c = 0; c < Columns; c++) {
			for (h = 0; h < KERNEL_SIZE; h++) {
				for (w = 0; w < KERNEL_SIZE; w++) {
					row_i = r - ((KERNEL_SIZE - 1) / 2) + h;
					col_i = c - ((KERNEL_SIZE - 1) / 2) + w;
					if ((row_i >= 0) && (row_i < Rows) && (col_i >= 0) && (col_i < Columns)) {
						results[r*Columns + c] += (M[h*KERNEL_SIZE + w] * N[row_i*Columns + col_i]);
					}
				}
			}
		}
	}

	equal = true;
	for (int i = 0; i < Rows * Columns && equal; i++) {
		if (abs(results[i] - P[i]) >= 0.001f) {
			equal = false;
			printf("NOT EQUAL!\n");
		}
	}

	if (equal) {
		printf("Results are equal!\n");
	}
	else {
		printf("Results are NOT equal!\n");
	}

	free(results);
	return;
}


void matrix_init_rand(float * matrix, int x, int y){
    int size = x*y;
	for(int i=0;i<size;i++){
		//matrix[i] = (float) ((rand()%1000) -500) / 100.0 ;
		matrix[i] = i; //for debug
	}
}

void print_matrix(float * matrix, int x, int y){
	for (int i=0;i<y;i++){
		for (int j=0;j<x;j++){
			printf("%f ", matrix[i*x + j]);	
		}
		printf("\n");
	}
}